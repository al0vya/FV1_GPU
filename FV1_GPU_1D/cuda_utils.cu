#include "cuda_utils.cuh"

void sync()
{
	CHECK_CUDA_ERROR( hipDeviceSynchronize() );
}

void peek()
{
	CHECK_CUDA_ERROR( hipPeekAtLastError() );
}

void reset()
{
	CHECK_CUDA_ERROR( hipDeviceReset() );
}

void copy
(
	void* dst,
	void* src,
	size_t bytes
)
{
	CHECK_CUDA_ERROR( hipMemcpy
	(
		dst,
		src,
		bytes,
		hipMemcpyDefault
	) );
}

void* malloc_device
(
	size_t bytes
)
{
	void* ptr;
	
	CHECK_CUDA_ERROR( hipMalloc
	(
		&ptr, 
		bytes
	) );

	return ptr;
}

void free_device
(
	void* ptr
)
{
	CHECK_CUDA_ERROR( hipFree(ptr) );
}